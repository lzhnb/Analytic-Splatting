#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "vec_math.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(
	const int idx,
	const int deg,
	const int max_coeffs,
	const glm::vec3* means,
	const glm::vec3 campos,
	const float* shs,
	bool* clamped
) {
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0) {
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1) {
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2) {
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
__device__ float3 computeCov2D(
	const float3& mean,
	float focal_x,
	float focal_y,
	float tan_fovx,
	float tan_fovy,
	const float* cov3D,
	const float* viewmatrix
) {
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	// NOTE: we only care the projected 2D cov, so we can ignore the third row in J
	// EWA splatting implement the standard camera coordinate in Eq. (29)
	// And we use the focal to scale the mat and make it suitable for our camera model
	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0.0f, 0.0f, 0.0f);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(
	const glm::vec3 scale,
	float mod,
	const glm::vec4 rot,
	float* cov3D
) {
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(
	const dim3 grid,
	const int P, int D, int M,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	const float* __restrict__ orig_points,
	const glm::vec3* __restrict__ scales,
	const float scale_modifier,
	const glm::vec4* __restrict__ rotations,
	const float* __restrict__ opacities,
	const float* __restrict__ shs,
	const float* __restrict__ cov3D_precomp,
	const float* __restrict__ colors_precomp,
	const float* __restrict__ viewmatrix,
	const float* __restrict__ projmatrix,
	const glm::vec3* __restrict__ cam_pos,
	int* __restrict__ radii,
	bool* __restrict__ clamped,
	float3* __restrict__ points_xy_image,
	float* __restrict__ depths,
	float* __restrict__ cov3Ds,
	float* __restrict__ rgb,
	float4* __restrict__ cov_opacity,
	float4* __restrict__ lambda_sigma,
	float4* __restrict__ nv1_nv2,
	uint32_t* __restrict__ tiles_touched,
	bool prefiltered
) {
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	const float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	const float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	const float p_w = 1.0f / (p_hom.w + 0.0000001f);
	const float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const float* cov3D;
	if (cov3D_precomp != nullptr) {
		cov3D = cov3D_precomp + idx * 6;
	} else {
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	const float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	const float det = cov.x * cov.z - cov.y * cov.y;
	if (det == 0.0f)
		return;

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	const float half_trace = 0.5f * (cov.x + cov.z);
	const float root = half_trace * half_trace - det;
	const float root_sqrt = sqrtf(max(0.0f, root));
	const float lambda1 = half_trace + root_sqrt;
	const float lambda2 = half_trace - root_sqrt;
	// const float lambda1_ = half_trace + max(0.33f, root_sqrt);
	// const float lambda2_ = half_trace - max(0.33f, root_sqrt);
	const float lambda1_ = half_trace + sqrt(max(0.1f, root));
	const float lambda2_ = half_trace - sqrt(max(0.1f, root));
	const float my_radius = ceil(3.f * sqrtf(max(lambda1_, lambda2_)));
	const float3 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H), root_sqrt};
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// https://math.stackexchange.com/questions/395698/fast-way-to-calculate-eigen-of-2x2-matrix-using-a-formula
	float2 v1 = {cov.y, lambda1 - cov.x};
	v1 = normalize(v1);
	float2 v2 = {lambda2 - cov.z, cov.y};
	v2 = normalize(v2);
	const float sigma1 = sqrtf(lambda1), sigma2 = sqrtf(abs(lambda2));

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr) {
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	radii[idx] = my_radius;
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	cov_opacity[idx] = { cov.x, cov.y, cov.z, opacities[idx] };
	lambda_sigma[idx] = {lambda1, lambda2, sigma1, sigma2};
	nv1_nv2[idx] = {v1.x, v1.y, v2.x, v2.y};
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const int W, int H,
	const float* __restrict__ bg_color,
	const float* __restrict__ features,
	const uint32_t* __restrict__ point_list,
	const float3* __restrict__ points_xy_image,
	const float4* __restrict__ cov_opacity,
	const float4* __restrict__ lambda_sigma,
	const float4* __restrict__ nv1_nv2,
	const uint2* __restrict__ ranges,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	float* __restrict__ out_color
) {
	// Identify current tile and associated min/max pixel range.
	const auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	const bool inside = pix.x < W && pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float3 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_cov_opacity[BLOCK_SIZE];
	__shared__ float4 collected_lambda_sigma[BLOCK_SIZE];
	__shared__ float4 collected_nv1_nv2[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0.0f };

	// Iterate over batches until all done or range is complete
    for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE) {
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y) {
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_cov_opacity[block.thread_rank()] = cov_opacity[coll_id];
			collected_lambda_sigma[block.thread_rank()] = lambda_sigma[coll_id];
			collected_nv1_nv2[block.thread_rank()] = nv1_nv2[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++) {
			// Keep track of current position in range
			contributor++;

			float3 xy = collected_xy[j];
			float2 d = { pixf.x - xy.x, pixf.y - xy.y };
			float4 cov_o = collected_cov_opacity[j];

			float2 v1 = {collected_nv1_nv2[j].x, collected_nv1_nv2[j].y}, v2 = {collected_nv1_nv2[j].z, collected_nv1_nv2[j].w};
			// // calculate the uv by projection
			float2 uv = {d.x * v1.x + d.y * v1.y, d.x * v2.x + d.y * v2.y};
			float sigma1 = collected_lambda_sigma[j].z, sigma2 = collected_lambda_sigma[j].w;

			// Equal to exp(power)
			// const float sigma1 = sqrtf(lambda1), sigma2 = sqrtf(abs(lambda2));
			const float U2 = (uv.x + 0.5f) / sigma1, U1 = (uv.x - 0.5f) / sigma1;
			const float cdfU2 = approxCdf(U2), cdfU1 = approxCdf(U1);
			const float intU = sigma1 * (cdfU2 - cdfU1);
			const float V2 = (uv.y + 0.5f) / sigma2, V1 = (uv.y - 0.5f) / sigma2;
			const float cdfV2 = approxCdf(V2), cdfV1 = approxCdf(V1);
			const float intV = sigma2 * (cdfV2 - cdfV1);
			const float integral = M_2PIf * intU * intV;

			float alpha = min(0.99f, cov_o.w * integral);

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			if (alpha < 1.0f / 255.0f) continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f) {
				done = true;
				continue;
			}

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside) {
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
	}
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const int W, int H,
	const float* bg_color,
	const float* colors,
	const uint32_t* point_list,
	const float3* means2D,
	const float4* cov_opacity,
	const float4* lambda_sigma,
	const float4* nv1_nv2,
	const uint2* ranges,
	float* final_T,
	uint32_t* n_contrib,
	float* out_color
) {
	renderCUDA<NUM_CHANNELS><<<grid, block>>>(
		W, H,
		bg_color,
		colors,
		point_list,
		means2D,
		cov_opacity,
		lambda_sigma,
		nv1_nv2,
		ranges,
		final_T,
		n_contrib,
		out_color);
}

void FORWARD::preprocess(
	const dim3 grid,
	const int P, int D, int M,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	int* radii,
	bool* clamped,
	float3* means2D,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* cov_opacity,
	float4* lambda_sigma,
	float4* nv1_nv2,
	uint32_t* tiles_touched,
	bool prefiltered
) {
	preprocessCUDA<NUM_CHANNELS><<<(P + 255) / 256, 256>>>(
		grid,
		P, D, M,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		radii,
		clamped,
		means2D,
		depths,
		cov3Ds,
		rgb,
		cov_opacity,
		lambda_sigma,
		nv1_nv2,
		tiles_touched,
		prefiltered
	);
}
